
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int64_t m, int64_t n,
          const hipComplex *alpha, const hipComplex *a, int64_t lda, hipComplex *b,
          int64_t ldb) {
  // Start
  hipblasCtrsm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
                 unit_diag /*cublasDiagType_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const cuComplex **/, a /*const cuComplex **/,
                 lda /*int64_t*/, b /*cuComplex **/, ldb /*int64_t*/);
  // End
}
