
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t op_a,
          hipsparseOperation_t op_b, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseSpMatDescr_t mat_b,
          const void *beta, hipsparseSpMatDescr_t mat_c,
          hipDataType compute_type, hipsparseSpGEMMAlg_t alg,
          hipsparseSpGEMMDescr_t desc, size_t *buffer_size, void *buffer) {
  // Start
  hipsparseSpGEMM_compute(
      handle /*cusparseHandle_t*/, op_a /*cusparseOperation_t*/,
      op_b /*cusparseOperation_t*/, alpha /*const void **/,
      mat_a /*cusparseSpMatDescr_t*/, mat_b /*cusparseSpMatDescr_t*/,
      beta /*const void **/, mat_c /*cusparseSpMatDescr_t*/,
      compute_type /*cudaDataType*/, alg /*cusparseSpGEMMAlg_t*/,
      desc /*cusparseSpGEMMDescr_t*/, buffer_size /*size_t **/,
      buffer /*void **/);
  // End
}
